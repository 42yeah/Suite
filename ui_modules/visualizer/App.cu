#include "hip/hip_runtime.h"
//
// Created by 42yea on 20/06/2022.
//

#include "App.cuh"
#include <glm/gtc/type_ptr.hpp>
#include <imgui.h>
#include <backends/imgui_impl_glfw.h>
#include <backends/imgui_impl_opengl3.h>
#include "ui_modules/primitives/prim.cuh"


constexpr int window_w = 1280 * 2, window_h = 720 * 2;


App::App() : window_size(0) {
    window = glfwCreateWindow(window_w, window_h, "Suite", nullptr, nullptr);
    glfwMakeContextCurrent(window);
    gladLoadGL();
    glClearColor(0.5f, 0.0f, 0.5f, 1.0f);
    glEnable(GL_DEPTH_TEST);

    program = std::make_shared<Program>("shaders/default/default.vert", "shaders/default/default.frag");
    bbox_program = std::make_shared<Program>("shaders/bbox/bbox.vert", "shaders/bbox/bbox.frag");
    camera = Camera(glm::vec3(0.0f, 0.0f, 5.0f), glm::radians(45.0f), (float) window_w / window_h, 0.1f, 100.0f);

    Scene scene_raw("models/ball.dae");
    scene = std::make_shared<SceneGL>(scene_raw);

    previous_instant = glfwGetTime();
    delta_time = 0.0f;

     ImGui::CreateContext();
     ImGuiIO &io = ImGui::GetIO();
     ImGui_ImplGlfw_InitForOpenGL(window, true);
     ImGui_ImplOpenGL3_Init("#version 330 core");
     io.IniFilename = nullptr;
     io.FontGlobalScale = 2.0f;
}

void App::run() {
    Program &prog = *this->program;

    while (!glfwWindowShouldClose(window)) {
        double time = glfwGetTime();
        delta_time = (float) (time - previous_instant);
        previous_instant = time;

        glfwPollEvents();
        update_camera();

        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        prog.use();
        scene->render_using(prog, camera);

        glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
        bbox_program->use();
        gen_bounding_box(scene->get_objects()[0].bbox()).render_using(*bbox_program, camera);
        glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);

        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::ShowDemoWindow();

        show_scripting_layer_window();

        ImGui::Render();
        glfwGetFramebufferSize(window, &window_size.x, &window_size.y);
        glViewport(0, 0, window_size.x, window_size.y);
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

        glfwSwapBuffers(window);
    }
}

App::~App() {
    glfwDestroyWindow(window);
}

void App::update_camera() {
    if (window_size.y != 0) {
        camera.aspect = (float) window_size.x / window_size.y;
    }

    glm::vec3 right = glm::normalize(glm::cross(camera.front, glm::vec3(0.0f, 1.0f, 0.0f)));
    if (glfwGetKey(window, GLFW_KEY_W)) {
        camera.eye += camera.front * camera.speed * delta_time;
    }
    if (glfwGetKey(window, GLFW_KEY_S)) {
        camera.eye -= camera.front * camera.speed * delta_time;
    }
    if (glfwGetKey(window, GLFW_KEY_D)) {
        camera.eye += right * camera.speed * delta_time;
    }
    if (glfwGetKey(window, GLFW_KEY_A)) {
        camera.eye -= right * camera.speed * delta_time;
    }

    if (glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_1)) {
        double xpos, ypos;
        glfwGetCursorPos(window, &xpos, &ypos);
        if (!start_cursor_pos) {
            start_cursor_pos = glm::vec2(xpos, ypos);
            start_py = glm::vec2(camera.pitch, camera.yaw);
        }

        glm::vec2 del_cursor_pos = glm::vec2(xpos, ypos) - *start_cursor_pos;
        del_cursor_pos = del_cursor_pos / (float) glm::max(window_w, window_h) * glm::pi<float>();
        camera.pitch = start_py->x - del_cursor_pos.y;
        camera.yaw = start_py->y + del_cursor_pos.x;
    } else if (start_cursor_pos) {
        // finalize
        start_cursor_pos = std::nullopt;
        start_py = std::nullopt;
    }

    camera.update_camera();
}

void App::show_scripting_layer_window() {
    static char buf[1024] = { 0 };
    ImGui::Begin("Lua Scripting Layer");

    // ImGui::SetNextItemWidth(-std::numeric_limits<float>::min());
    ImGui::InputText("command", buf, sizeof(buf));

    ImGui::SameLine();

    if (ImGui::Button("OK")) {
        layer(buf);
    }

    ImGui::BeginTable("error_list", 1);
    for (const auto &err : layer.get_errors()) {
        ImGui::TableNextRow();
        ImGui::TableNextColumn();

        ImGui::Selectable(err.c_str());
    }
    ImGui::EndTable();

    ImGui::End();
}

void start_app() {
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    App app;
    app.run();
}
