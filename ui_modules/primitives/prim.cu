#include "hip/hip_runtime.h"
//
// Created by 42yea on 2022/6/22.
//

#include "prim.cuh"
#include <glm/glm.hpp>


std::shared_ptr<ModelGL> gen_triangle() {
    std::vector<Vertex> vertices;

    vertices.push_back(Vertex{
        glm::vec3(0.0f, 0.0f, 0.0f),
        glm::vec3(0.0f, 0.0f, 1.0f),
        glm::vec2(0.0f, 0.0f)
    });
    vertices.push_back(Vertex{
            glm::vec3(0.5f, 0.0f, 0.0f),
            glm::vec3(0.0f, 0.0f, 1.0f),
            glm::vec2(0.5f, 0.0f)
    });
    vertices.push_back(Vertex{
            glm::vec3(0.0f, 0.5f, 0.0f),
            glm::vec3(0.0f, 0.0f, 1.0f),
            glm::vec2(0.0f, 0.5f)
    });

    Model model(vertices);
    std::shared_ptr<ModelGL> model_gl = std::make_shared<ModelGL>(model);
    return model_gl;
}

std::shared_ptr<ModelGL> box = nullptr;

ObjectGL gen_bounding_box(const BBox &bbox) {
    if (box == nullptr) {
        glm::vec3 v0(-1.0f, -1.0f, -1.0f),
            v1(1.0f, -1.0f, -1.0f),
            v2(1.0f, -1.0f, 1.0f), //
            v3(-1.0f, -1.0f, 1.0f), //
            v4(-1.0f, 1.0f, -1.0f),
            v5(1.0f, 1.0f, -1.0f),
            v6(1.0f, 1.0f, 1.0f), //
            v7(-1.0f, 1.0f, 1.0f); //

        glm::vec3 n0(0.0f, -1.0f, 0.0f),
            n1(0.0f, 1.0f, 0.0f),
            n2(-1.0f, 0.0f, 0.0f),
            n3(1.0f, 0.0f, 0.0f),
            n4(0.0f, 0.0f, -1.0f),
            n5(0.0f, 0.0f, 1.0f);

        glm::vec2 t0(0.0f, 0.0f),
            t1(1.0f, 0.0f),
            t2(1.0f, 1.0f),
            t3(0.0f, 1.0f);

        std::vector<Vertex> vertices = {
            Vertex{ v0, n0, t0 },
            Vertex{ v1, n0, t1 },
            Vertex{ v2, n0, t2 },
            Vertex{ v2, n0, t2 },
            Vertex{ v3, n0, t3 },
            Vertex{ v0, n0, t0 },

            Vertex{ v4, n1, t0 },
            Vertex{ v5, n1, t1 },
            Vertex{ v6, n1, t2 },
            Vertex{ v6, n1, t2 },
            Vertex{ v7, n1, t3 },
            Vertex{ v4, n1, t0 },

            Vertex{ v0, n2, t0 },
            Vertex{ v3, n2, t1 },
            Vertex{ v7, n2, t2 },
            Vertex{ v7, n2, t2 },
            Vertex{ v4, n2, t3 },
            Vertex{ v0, n2, t0 },

            Vertex{ v1, n3, t0 },
            Vertex{ v2, n3, t1 },
            Vertex{ v6, n3, t2 },
            Vertex{ v6, n3, t2 },
            Vertex{ v5, n3, t3 },
            Vertex{ v1, n3, t0 },

            Vertex{ v0, n4, t0 },
            Vertex{ v1, n4, t1 },
            Vertex{ v5, n4, t2 },
            Vertex{ v5, n4, t2 },
            Vertex{ v4, n4, t3 },
            Vertex{ v0, n4, t0 },

            Vertex{ v3, n5, t0 },
            Vertex{ v2, n5, t1 },
            Vertex{ v6, n5, t2 },
            Vertex{ v6, n5, t2 },
            Vertex{ v7, n5, t3 },
            Vertex{ v3, n5, t0 }
        };

        Model model(vertices);
        box = std::make_shared<ModelGL>(model);
    }

    glm::vec3 box_center = (bbox.min + bbox.max) / 2.0f;
    glm::mat4 transform = glm::translate(glm::mat4(1.0f), box_center);
    transform = glm::scale(transform, bbox.span() * 0.5f);
    return { box, transform };
}
