#include "hip/hip_runtime.h"
//
// Created by 42yea on 2022/6/21.
//

#include "ObjectGL.cuh"


ObjectGL::ObjectGL() : model(nullptr), transform(1.0f) {

}

ObjectGL::ObjectGL(std::shared_ptr<ModelGL> model, glm::mat4 transform) : model(std::move(model)), transform(transform) {

}

void ObjectGL::render_using(Program &program, const Camera &camera) const {
    glUniformMatrix4fv(program["model"], 1, GL_FALSE, glm::value_ptr(transform));
    glUniformMatrix4fv(program["view"], 1, GL_FALSE, glm::value_ptr(camera.view));
    glUniformMatrix4fv(program["perspective"], 1, GL_FALSE, glm::value_ptr(camera.perspective));
    if (model == nullptr) {
        std::cerr << "WARNING! Null model detected; that's are not supposed to happen." << std::endl;
    }
    model->render_using(program);
}

BBox ObjectGL::bbox() const {
    BBox bbox = model->get_bbox();
    bbox.min = glm::vec3(transform * glm::vec4(bbox.min, 1.0f));
    bbox.max = glm::vec3(transform * glm::vec4(bbox.max, 1.0f));
    return bbox;
}

