#include "hip/hip_runtime.h"
//
// Created by 42yea on 2022/6/21.
//

#include "Program.cuh"
#include <fstream>
#include <sstream>
#include <optional>


std::optional<GLuint> compile(GLuint type, const std::string &path) {
    std::ifstream reader(path);

    if (!reader.good()) {
        std::cerr << "ERR! Cannot open: " << path << std::endl;
        return std::nullopt;
    }
    std::stringstream ss;
    ss << reader.rdbuf();
    std::string str = ss.str();
    const char *src = str.c_str();

    GLuint shader = glCreateShader(type);
    glShaderSource(shader, 1, &src, nullptr);
    glCompileShader(shader);

    GLint state;
    glGetShaderiv(shader, GL_COMPILE_STATUS, &state);
    if (state != GL_TRUE) {
        char log[512] = { 0 };
        glGetShaderInfoLog(shader, sizeof(log), nullptr, log);
        std::cerr << "ERR! Failed to ocmpile shader: " << path << ": " << log << std::endl;
        glDeleteShader(shader);
        return std::nullopt;
    }
    return shader;
}

std::optional<GLuint> link(GLuint vertex_shader, GLuint fragment_shader) {
    GLuint program = glCreateProgram();
    glAttachShader(program, vertex_shader);
    glAttachShader(program, fragment_shader);
    glLinkProgram(program);
    GLint state;
    glGetProgramiv(program, GL_LINK_STATUS, &state);
    if (state != GL_TRUE) {
        char log[512] = { 0 };
        glGetProgramInfoLog(program, sizeof(log), nullptr, log);
        std::cerr << "ERR! Failed to link shader: " << log << std::endl;
        glDeleteProgram(program);
        return std::nullopt;
    }
    return program;
}

Program::Program(const std::string &vertex_shader_path, const std::string &fragment_shader_path) :
    program(0) {

    GLuint vertex_shader = *compile(GL_VERTEX_SHADER, vertex_shader_path);
    GLuint fragment_shader = *compile(GL_FRAGMENT_SHADER, fragment_shader_path);

    program = *link(vertex_shader, fragment_shader);

    glDeleteShader(vertex_shader);
    glDeleteShader(fragment_shader);
}

GLint Program::operator[](const std::string &uniform_name) {
    if (loc.find(uniform_name) == loc.end()) {
        GLint pos = glGetUniformLocation(program, uniform_name.c_str());
        if (pos < 0) {
            std::cerr << "WARNING! Cannot find uniform: " << uniform_name << std::endl;
        }
        loc[uniform_name] = pos;
    }
    return loc[uniform_name];
}

Program::~Program() {
    if (program != 0) {
        glDeleteProgram(program);
    }
}
