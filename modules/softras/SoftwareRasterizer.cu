#include "hip/hip_runtime.h"
//
// Created by 42yea on 29/06/2022.
//

#include "SoftwareRasterizer.cuh"
#include <vector>
#include <hip/hip_runtime.h>
#include <>
#include "modules/model/Model.cuh"
#include "modules/model/Vertex.cuh"
#include "modules/geoquery/Triangle.cuh"
#include "modules/cuda/CudaPtr.cuh"


const Texture &SoftwareRasterizer::render(const Scene &scene, const Camera &camera) {
    // for each object...
    for (const Object &obj : scene.get_objects()) {
        rasterize(obj, output, camera);
    }
    return output;
}

const Texture &SoftwareRasterizer::render(const Object &object, const Camera &camera) {
    rasterize(object, output, camera);
    return output;
}

SoftwareRasterizer::SoftwareRasterizer(glm::ivec2 size) : size(size), output(size), depth(size) {
    output.clear(glm::vec4(0.5f, 0.0f, 0.5f, 1.0f));
    depth.clear(glm::vec4(std::numeric_limits<float>::max(), 0.0f, 0.0f, 1.0f));
}

__global__ void software_vertex_shader(glm::mat4 model, glm::mat4 view, glm::mat4 perspective,
                                       Vertex *vertices, Triangle *triangles, int n_triangles) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x > n_triangles) {
        return;
    }

    for (int i = 0; i < 3; i++) {
        Vertex vert = vertices[x * 3 + i];

        vert.normal = glm::vec3(model * glm::vec4(vert.normal, 0.0f));

        // perform perspective divide
        glm::vec4 pos = perspective * view * model * glm::vec4(vert.position, 1.0f);
        vert.position = glm::vec3(pos) / pos.w;

        switch (i) {
            case 0:
                triangles[x].a = vert;
                break;

            case 1:
                triangles[x].b = vert;
                break;

            case 2:
                triangles[x].c = vert;
                break;

            default:
                break;
        }

        triangles[x].bbox.enclose(vert.position);
        triangles[x].center += vert.position;

    }

    triangles[x].center /= 3.0f;

}

__global__ void software_fragment_shader(Triangle *triangles, int n_tri, glm::vec4 *tex, glm::vec4 *depth,
                                         glm::ivec2 size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > n_tri) {
        return;
    }

    const Triangle &tri = triangles[i];
    BBox bbox = tri.bbox;

    if ((bbox.min.x > 1.0f || bbox.min.y > 1.0f) || (bbox.max.x < -1.0f || bbox.max.y < -1.0f)) {
        return;
    }

    Triangle tri_screen(glm::vec3(0.5f * tri.a.position + 0.5f) * glm::vec3(size, 0.0f),
            glm::vec3(0.5f * tri.b.position + 0.5f) * glm::vec3(size, 0.0f),
            glm::vec3(0.5f * tri.c.position + 0.5f) * glm::vec3(size, 0.0f));

    BBox bbox_screen = tri_screen.bbox;

    glm::ivec2 screen_min = glm::max(glm::ivec2(bbox_screen.min), 0);
    glm::ivec2 screen_max = glm::min(glm::ivec2(bbox_screen.max), glm::ivec2(size.x - 1, size.y - 1));

    for (int y = screen_min.y; y <= screen_max.y; y++) {
        for (int x = screen_min.x; x <= screen_max.x; x++) {
            // perform barycentric triangle test
            glm::vec3 barycentric = tri_screen.barycentric(glm::vec3(x, y, 0.0f));
            if (barycentric.x < 0.0f || barycentric.y < 0.0f || barycentric.z < 0.0f) {
                continue;
            }
            // barycentric interpolate current depth value & perform depth testing
            float z = barycentric.x * tri.a.position.z + barycentric.y * tri.b.position.z + barycentric.z * tri.c.position.z;
            if (depth[y * size.x + x].r < z) {
                continue;
            }
            depth[y * size.x + x].r = z;

            // evaluate phong shading
            glm::vec3 normal = barycentric.x * tri.a.normal + barycentric.y * tri.b.normal +
                               barycentric.z * tri.c.normal;
            glm::vec3 obj_color = glm::vec3(1.0f, 1.0f, 1.0f);
            float ambient = 0.2f;
            float diffuse = glm::max(glm::dot(normal, glm::vec3(0.0f, 1.0f, 0.0f)), 0.0f);

            tex[y * size.x + x] = glm::vec4((ambient + diffuse) * obj_color, 1.0f);
        }
    }
}

void SoftwareRasterizer::rasterize(const Object &object, Texture &texture, const Camera &camera) {
    std::vector<Vertex> vertices = object.model->get_vertices();
    std::vector<Triangle> triangles;
    triangles.resize(vertices.size() / 3);

    {
        CudaPtr<glm::vec4> tex_data(&texture(0, 0), texture.get_size().x * texture.get_size().y);
        CudaPtr<glm::vec4> depth_data(&depth(0, 0), depth.get_size().x * depth.get_size().y);
        CudaPtr<Vertex> vertices_cuda(&vertices[0], vertices.size());
        CudaPtr<Triangle> triangles_cuda(&triangles[0], triangles.size());

        constexpr int block_size = 512;
        int n_blocks = triangles.size() / block_size + 1;
        software_vertex_shader<<<n_blocks, block_size>>>(object.transform,
                                                         camera.view, camera.perspective,
                                                         vertices_cuda.get(),
                                                         triangles_cuda.get(),
                                                         triangles.size());

        // rasterize each triangle
        software_fragment_shader<<<n_blocks, block_size>>>(triangles_cuda.get(),
                                                           triangles.size(),
                                                           tex_data.get(),
                                                           depth_data.get(),
                                                           texture.get_size());
    }
}

void SoftwareRasterizer::clear(glm::vec4 color) {
    output.clear(color);
    depth.clear(glm::vec4(std::numeric_limits<float>::max(), 0.0f, 0.0f, 1.0f));
}

