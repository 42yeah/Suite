//
// Created by 42yea on 21/06/2022.
//

#include "Model.cuh"


Model::Model(std::vector<Vertex> m_vertices) : vertices(std::move(m_vertices)), center(0.0f) {
    for (const auto &v : vertices) {
        center += v.position;
        bbox.enclose(v.position);
    }
    center /= vertices.size();
}

const std::vector<Vertex> &Model::get_vertices() const {
    return vertices;
}

glm::vec3 Model::get_center() const {
    return center;
}

BBox Model::get_bbox() const {
    return bbox;
}
