#include "hip/hip_runtime.h"
//
// Created by 42yea on 29/06/2022.
//

#include "Texture.cuh"
#include <hip/hip_runtime.h>
#include <>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "deps/stb/stb_image_write.h"
#include "modules/cuda/CudaPtr.cuh"


__global__ void clear_texture_cuda(glm::vec4 *what, glm::ivec2 size, glm::vec4 color) {
    int x = threadIdx.x, y = blockIdx.x;

    what[y * size.x + x] = color;
}

__global__ void standardize_texture(glm::vec4 *input, glm::u8vec3 *output, glm::ivec2 size) {
    int x = threadIdx.x, y = blockIdx.x;

    output[y * size.x + x].r = (unsigned char) glm::clamp(input[y * size.x + x].r * 255.0f, 0.0f, 255.0f);
    output[y * size.x + x].g = (unsigned char) glm::clamp(input[y * size.x + x].g * 255.0f, 0.0f, 255.0f);
    output[y * size.x + x].b = (unsigned char) glm::clamp(input[y * size.x + x].b * 255.0f, 0.0f, 255.0f);
}


Texture::Texture(glm::ivec2 size) : size(size), data(new glm::vec4[size.x * size.y]) {

}

void Texture::clear(glm::vec4 color) {
    CudaPtr<glm::vec4> cuda_ptr(data.get(), size.x * size.y);

    clear_texture_cuda<<<size.y, size.x>>>(cuda_ptr.get(), size, color);
}

void Texture::export_as_jpg(const std::filesystem::path &path) const {
    std::shared_ptr<glm::u8vec3[]> standardized(new glm::u8vec3[size.x * size.y]);
    CudaPtr<glm::vec4> input_cuda(data.get(), size.x * size.y);

    {
        CudaPtr<glm::u8vec3> output_cuda(standardized.get(), size.x * size.y);

        standardize_texture<<<size.y, size.x>>>(input_cuda.get(), output_cuda.get(), size);
    }

    stbi__flip_vertically_on_write = true;
    stbi_write_jpg(path.string().c_str(), size.x, size.y, 3, standardized.get(), 100);
}

glm::vec4 &Texture::operator()(int x, int y) {
    if (x < 0 || x > size.x || y < 0 || y > size.y) {
        throw std::exception("Texture indexing out of bounds");
    }
    return data[y * size.x + x];
}

glm::ivec2 Texture::get_size() const {
    return size;
}
